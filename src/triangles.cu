
#include <hip/hip_runtime.h>
/* 
CUDA code to find perfect and imperfect triangles for dual-types. 
~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
Why does a GPU need to be used for this? 
Mathematical Theory:
~~~~~~~~~~~~~~~~~~~~
There are 18 mono-types in Pokemon. They are enumerated, and their damage relationships implemented in "../include/SusceptGraph.hpp" 

Each of these 18 mono-types has 17 possible mono-types that it can be paired with, giving 18*17 = 306 possible dual-types. 

Together, there are 18 + 306 = 324 possible type combinations that a Pokemon can have. 

However, type combinations are associative, i.e., dual-type AB is basically the same as dual-type BA.

Therefore, there are 18 + 306 / 2 = 18 + 153 = 171 unique possible type combinations that a pokemon can have.  

This order-of-magnitude increase in the number of types significantly increases the complexity of the problem, and the additional structure from the secondary 
type requires that a second look be taken at exactly *what* we are searching for. 

We want to find "perfect", and "imperfect", triangles amongst the possible dual-types.
*/